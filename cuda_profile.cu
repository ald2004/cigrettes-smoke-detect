
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>
// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
//#if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n",
            hipGetErrorString(result));
        assert(result == hipSuccess);
    }
//#endif
    return result;
}
static hipStream_t streamsArray[16];    // cudaStreamSynchronize( get_cuda_stream() );
static int streamInit[16] = { 0 };
void profileCopies(float* h_a,
    float* h_b,
    float* d,
    unsigned int  n,
    char* desc)
{
    printf("\n%s transfers\n", desc);

    unsigned int bytes = n * sizeof(float);

    // events for timing
    hipEvent_t startEvent, stopEvent;

    checkCuda(hipEventCreate(&startEvent));
    checkCuda(hipEventCreate(&stopEvent));

    checkCuda(hipEventRecord(startEvent, 0));
    checkCuda(hipMemcpy(d, h_a, bytes, hipMemcpyHostToDevice));
    checkCuda(hipEventRecord(stopEvent, 0));
    checkCuda(hipEventSynchronize(stopEvent));

    float time;
    checkCuda(hipEventElapsedTime(&time, startEvent, stopEvent));
    printf("  Host to Device bandwidth (GB/s): %f\n", bytes * 1e-6 / time);

    checkCuda(hipEventRecord(startEvent, 0));
    checkCuda(hipMemcpy(h_b, d, bytes, hipMemcpyDeviceToHost));
    checkCuda(hipEventRecord(stopEvent, 0));
    checkCuda(hipEventSynchronize(stopEvent));

    checkCuda(hipEventElapsedTime(&time, startEvent, stopEvent));
    printf("  Device to Host bandwidth (GB/s): %f\n", bytes * 1e-6 / time);

    for (int i = 0; i < n; ++i) {
        if (h_a[i] != h_b[i]) {
            printf("*** %s transfers failed ***\n", desc);
            break;
        }
    }

    // clean up events
    checkCuda(hipEventDestroy(startEvent));
    checkCuda(hipEventDestroy(stopEvent));
}
int cuda_get_device()
{
    int n = 0;
    hipError_t status = hipGetDevice(&n);
    checkCuda(status);
    return n;
}
hipStream_t get_cuda_stream() {
    int i = cuda_get_device();
    if (!streamInit[i]) {
        //printf("Create CUDA-stream \n");
        hipError_t status = hipStreamCreate(&streamsArray[i]);
        //cudaError_t status = cudaStreamCreateWithFlags(&streamsArray[i], cudaStreamNonBlocking);
        if (status != hipSuccess) {
            printf(" hipStreamCreate error: %d \n", status);
            const char* s = hipGetErrorString(status);
            printf("CUDA Error: %s\n", s);
            status = hipStreamCreateWithFlags(&streamsArray[i], hipStreamDefault);
            checkCuda(status);
        }
        streamInit[i] = 1;
    }
    return streamsArray[i];
}


void error(const char* s)
{
    perror(s);
    assert(0);
    exit(EXIT_FAILURE);
}
float* cuda_make_array(float* x, size_t n)
{
    float* x_gpu;
    size_t size = sizeof(float) * n;
    hipError_t status = hipMalloc((void**)&x_gpu, size);
    //cudaError_t status = cudaMallocManaged((void **)&x_gpu, size, cudaMemAttachGlobal);
    //status = cudaMemAdvise(x_gpu, size, cudaMemAdviseSetPreferredLocation, cudaCpuDeviceId);
    if (status != hipSuccess) fprintf(stderr, " Try to set subdivisions=64 in your cfg-file. \n");
    checkCuda(status);
    if (x) {
        //status = cudaMemcpy(x_gpu, x, size, cudaMemcpyHostToDevice);
        status = hipMemcpyAsync(x_gpu, x, size, hipMemcpyDefault, get_cuda_stream());
        checkCuda(status);
    }
    if (!x_gpu) error("Cuda malloc failed\n");
    return x_gpu;
}
int main()
{
    int n;
    hipDeviceProp_t prop;
    
    unsigned int nElements = 4 * 1024 * 1024;
    const unsigned int bytes = nElements * sizeof(float);

    
    hipError_t status;
    int a[] = { 0, 1 };
    for (auto i : a) {
        checkCuda(hipGetDeviceProperties(&prop, i));
        printf("\nDevice: %s\n", prop.name);
        printf("Transfer size (MB): %d\n", bytes / (1024 * 1024));
        status = hipSetDevice(i);
        checkCuda(status);
        status = hipGetDevice(&n);
        checkCuda(status);
        status = hipStreamCreate(&streamsArray[i]);
        checkCuda(status);
        status = hipStreamCreate(&streamsArray[i]);
        checkCuda(status);

        float* xx=cuda_make_array(0, 4435968);
        printf("%f ", xx);
        
    }
    getchar();
    return 0;
}
